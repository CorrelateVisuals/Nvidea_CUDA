﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>

// Device code
__global__ void MyKernel(int* d, int* a, int* b)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d[idx] = a[idx] * b[idx];
}

// Host code
int main()
{
    int numBlocks;        // Occupancy in terms of active blocks
    int blockSize = 64;

    // These variables are used to convert occupancy to warps
    int device;
    hipDeviceProp_t prop;
    int activeWarps;
    int maxWarps;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        MyKernel,
        blockSize,
        0);

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;

    return 0;
}